#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C"{
#endif

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "NengoGPU.h"
#include "NengoGPU_CUDA.h"

# define MAX_SHARED_MEM_SIZE 16000

// print the contents of an array of integers located on the device
void printIntArrayFromDevice(FILE* fp, intArray* a, jint n, jint m, jint labels)
{
  jint* temp = (jint*) malloc( m * n * sizeof(jint));
  hipMemcpy(temp, a->array, m * n * sizeof(jint), hipMemcpyDeviceToHost);

  printf("%s:\n", a->name);

  jint i, j;
  for(i = 0; i < n; i++)
  {
    fp ? fprintf(fp, "line %d: ", i) : printf("line %d:", i);
    for(j = 0; j < m; j++)
    {
      if(labels)
        fp ? fprintf(fp, "(%d, %d) ", j, temp[i * n + j]) : printf("(%d, %d) ", j, temp[i * n + j]);
      else
        fp ? fprintf(fp, "%d ", temp[i * n + j]) : printf("%d ", temp[i * n + j]);
    }
    fp ? fprintf(fp, "\n") : printf("\n");
  }

  fp ? fprintf(fp, "\n") : printf("\n");

  free(temp);
}

// print the contents of an array of floats located on the device
void printFloatArrayFromDevice(FILE* fp, floatArray* a, jint n, jint m, jint labels)
{
  hipError_t err;
  float* temp = (float*) malloc( m * n * sizeof(float));
  err = hipMemcpy(temp, a->array, m * n * sizeof(float), hipMemcpyDeviceToHost);
  checkCudaError(err, "in printFloatArrayFromDevice, copying from device to host");

  printf("%s:\n", a->name);

  jint i, j;
  for(i = 0; i < n; i++)
  {
    fp ? fprintf(fp, "line %d: ", i) : printf("line %d:", i);
    for(j = 0; j < m; j++)
    {
      if(labels)
        fp ? fprintf(fp, "(%d, %f) ", j, temp[i * n + j]) : printf("(%d, %f) ", j, temp[i * n + j]);
      else
        fp ? fprintf(fp, "%f ", temp[i * n + j]) : printf("%f ", temp[i * n + j]);
    }

    fp ? fprintf(fp, "\n") : printf("\n");
  }

  fp ? fprintf(fp, "\n") : printf("\n");

  free(temp);
}

void printIntColumn(FILE* fp, jint* array, jint m, jint n, jint col)
{
  jint* temp = (jint*) malloc( m * n * sizeof(jint));
  hipMemcpy(temp, array, m * n * sizeof(jint), hipMemcpyDeviceToHost);

  jint i;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "%d ", temp[i * n + col]) : printf("%d ", temp[i * n + col]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

void printFloatColumn(FILE* fp, float* array, jint m, jint n, jint col)
{
  float* temp = (float*) malloc( m * n * sizeof(float));
  hipMemcpy(temp, array, m * n * sizeof(float), hipMemcpyDeviceToHost);

  jint i;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "%f ", temp[i * n + col]) : printf("%f ", temp[i * n + col]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}
 
void printFloatRange(FILE* fp, float* array, jint start, jint end)
{
  float* temp = (float*) malloc((end - start + 1)  * sizeof(float));
  hipMemcpy(temp, array + start, (end - start + 1) * sizeof(float), hipMemcpyDeviceToHost);

  jint i;
  for(i = 0; i < end - start + 1; i++)
  {
    fp ? fprintf(fp, "%f ", temp[i]) : printf("%f ", temp[i]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

void printIntRange(FILE* fp, jint* array, jint start, jint end)
{
  jint* temp = (jint*) malloc((end - start + 1)  * sizeof(jint));
  hipMemcpy(temp, array + start, (end - start + 1) * sizeof(jint), hipMemcpyDeviceToHost);

  jint i;
  for(i = 0; i < end - start + 1; i++)
  {
    fp ? fprintf(fp, "%d ", temp[i]) : printf("%d ", temp[i]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

// get number of devices available
int getGPUDeviceCount(){
  //hipError_t err;
  int numDevices;
  
  hipGetDeviceCount(&numDevices);
  //checkCudaError(err, "get GPU device count");
  
  return numDevices;
}

// Reserves device with number deviceNum for the thread that calls this function. No interaction with the device should take place until this has been called.
// Once the device is reserved for the thread, no other thread should try to interact with that device or reserve it. A thread can reserve only one device at a time
void initGPUDevice(jint deviceNum)
{
  hipSetDevice(deviceNum);
}

void shutdownGPUDevice()
{
}

void checkCudaErrorWithDevice(hipError_t err, jint device, char* message)
{
  if(!err)
      return;

  printf("device: %d", device);
  checkCudaError(err, message);
}

void checkCudaError(hipError_t err, char* message)
{
    if(!err)
        return;

    printf(" CUDA ERROR: message: %s, description: %s\n", message, hipGetErrorString(err));

    exit(EXIT_FAILURE);
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalNumTerminationRows kernels are launched.
// Dot product the ith termination row with the corresponding input vector. Integrate the result. Results are stored in terminationValues. 
__global__ void transform(float dt, jint numTransformRows, float* input, jint* terminationOffsetInInput, jint* transformRowToInputIndexor, float* transforms, float* tau, float* terminationOutput, jint* terminationOutputIndexor, jint* inputDimensions)
{
  
  jint i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < numTransformRows)
  {
    
    jint j;
    jint inputIndex = transformRowToInputIndexor[i];
    jint offset = terminationOffsetInInput[inputIndex];
    
    jint inputDimension = inputDimensions[inputIndex];
    jint transformRowIndex = i;
    
    float my_tau = tau[inputIndex];
    
    float dot_product = 0;
    
    for(j=0; j < inputDimension; j++)
    {
      dot_product += input[offset + j] * transforms[transformRowIndex];

      transformRowIndex += numTransformRows;
    }
   
    float dt_over_tau = dt / my_tau;
    

    jint outputIndex = terminationOutputIndexor[i];
    terminationOutput[outputIndex] = (1 - dt_over_tau) * terminationOutput[outputIndex] + dt_over_tau * dot_product;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalDimension kernels are launched.
// Sum the termination values for one dimension of one ensemble. Results are stored in ensembleSums.
__global__ void sumTerminations(jint totalDimensions, jint maxNumDecodedTerminations, float* terminationOutput, float* ensembleSums)
{
  jint i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < totalDimensions)
  {
    jint terminationOutputIndex = i;
    jint j;
    float sum = 0;

    for(j=0; j < maxNumDecodedTerminations; j++)
    {
      sum += terminationOutput[terminationOutputIndex];
      terminationOutputIndex += totalDimensions;
    }

    ensembleSums[i] = sum;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least numNeurons kernels are launched.
// Multiply one encoder row by the sum vector for the corresponding ensemble. Then integrate to determine whether the neuron corresponding to that encoder row should spike. Results stored in spikes.
__global__ void encode(jint totalNumNeurons, float* encoders, float* sums, float* encodeResult, jint* encoderRowToEnsembleIndexor, jint* ensembleOffsetInDimension, jint* ensembleDimension, jint* encoderStride, jint* neuronIndexor)
{
  jint i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < totalNumNeurons)
  {
    jint ensembleIndex = encoderRowToEnsembleIndexor[i];
    jint currentEnsembleDimension = ensembleDimension[ensembleIndex];
    jint dimensionOffset = ensembleOffsetInDimension[ensembleIndex];

    jint j, encoderOffset = i;
    float dot_product = 0;


    for(j = 0; j < currentEnsembleDimension; j++)
    {
      dot_product += encoders[encoderOffset] * sums[dimensionOffset + j];
      encoderOffset += encoderStride[j];
    }
    
    jint neuronIndex = neuronIndexor[i];
    encodeResult[neuronIndex] = dot_product;
  }
}

__global__ void integrateAfterEncode(jint numNeurons, float dt, float adjusted_dt, jint steps, jint* neuronToEnsembleIndexor, float* encodingResult, float* neuronVoltage, float* neuronReftime, float* tau_RC, float* tauRef, float* bias, float* scale, float* spikes, float* NDterminationSums, jint* isSpikingEnsemble)
{
  jint i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;
  
  if( i < numNeurons)
  {
    jint ensembleIndex = neuronToEnsembleIndexor[i];
    float voltage = neuronVoltage[i];
    float refTime = neuronReftime[i];
    float tau_rc = tau_RC[ensembleIndex];
    float tau_ref = tauRef[ensembleIndex];
    float current = bias[i] + scale[i] * (encodingResult[i] + NDterminationSums[ensembleIndex]);

    if(isSpikingEnsemble[ensembleIndex])
    {
      float dV, post_ref, v_threshold = 1.0f;
      float spike_float;
      jint j, spike = 0;

      for(j = 0; j < steps; j++)
      {
        dV = adjusted_dt / tau_rc * (current - voltage);
        voltage = max(voltage + dV, 0.0f);

        post_ref = 1.0f - (refTime - adjusted_dt) / adjusted_dt;

        voltage = post_ref >= 1.0f ? voltage : voltage * post_ref;

        voltage = post_ref <= 0.0f ? 0.0f : voltage;

        spike = spike ? spike : voltage > v_threshold;
        spike_float = spike ? 1.0f/dt : 0.0f;
        refTime = spike ? ((adjusted_dt / dV) * (dV - voltage + v_threshold)) + tau_ref : refTime - adjusted_dt;
        voltage = spike ? 0.0 : voltage;
      }

      neuronReftime[i] = refTime;
      neuronVoltage[i] = voltage;
      spikes[i] = spike_float;
    }
    else
    {
      spikes[i] = (current > 1.0) ? 1.0 / (tau_ref - tau_rc * logf(1.0 - 1.0 / current)) : 0.0;
    }
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalOutputSize kernels are launched.
// Multiply one decoder row by the spike vector for the corresponding ensemble. The result is one dimension of the output vector for the ensemble. Results stored in output.
__global__ void decode(jint totalOutputSize, float* decoders, float* spikes, float* output, jint* decoderRowToEnsembleIndexor, jint* ensembleNumNeurons, jint* ensembleOffsetInNeurons, jint* decoderStride, jint* outputIndexor)
{
  jint i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;
  
  if( i < totalOutputSize)
  {
    
    jint ensembleIndex = decoderRowToEnsembleIndexor[i];
    jint numNeurons = ensembleNumNeurons[ensembleIndex];
    jint spikesOffset = ensembleOffsetInNeurons[ensembleIndex];
    
    jint j, decoderOffset = i;
    float dot_product = 0;

    for(j=0; j < numNeurons; j++)
    {
        dot_product += decoders[decoderOffset] * spikes[spikesOffset + j];

        decoderOffset += decoderStride[j];
    }
    

    jint currentOutputIndex = outputIndexor[i];
    output[currentOutputIndex] = dot_product;
  }
}



// launch as many as there are ensembles
__global__ void processNDterminations(jint numEnsembles, jint numNDterminations, jint steps, float adjusted_dt, jint* NDterminationEnsembleOffset, jint* terminationOffsetInInputs, jint* terminationDimensions, jint* inputIndex, float* input, float* weights, float* current, float* sum, float* tau)
{
  jint i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < numEnsembles)
  {
    jint offset = NDterminationEnsembleOffset[i];
    jint count = (i == numEnsembles - 1) ? numNDterminations - offset : NDterminationEnsembleOffset[i+1] - offset;
    jint j, k, terminationOffsetInInput, terminationDimension, index;
    float val, temp_sum = 0, temp_current, temp_tau;

    jint weightIndexInEnsemble = i;

    if(count > 0)
    {
      for(j = 0; j < count; j++)
      {
        index = inputIndex[offset + j];
        terminationOffsetInInput = terminationOffsetInInputs[index]; 
        terminationDimension = terminationDimensions[index];

        val = 0;
        for(k = 0; k < terminationDimension; k++)
        {
          // have to figure out how to index this properly
          val += input[terminationOffsetInInput + k] * weights[weightIndexInEnsemble];
          weightIndexInEnsemble += numEnsembles;
        }

        temp_current = current[offset + j];
        temp_tau = tau[index];

        for(k = 0; k < steps; k++)
        {
          // testing this order, though this is the one used in the java code so it should work
          temp_current *= 1 - adjusted_dt / temp_tau;
          temp_current += val * adjusted_dt / temp_tau;
        }

        current[offset + j] = temp_current;
        
        temp_sum += temp_current;
      }

      sum[i] = temp_sum;
    }
  }
}


__global__ void moveGPUData(jint size, jint* map, float* to, float* from)
{
  jint i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < size)
  {
    to[i] = from[ map[i] ];
  }
}
      
// run a NengoGPUData struct for one step
void run_NEFEnsembles(NengoGPUData* nengoData, float startTime, float endTime)
{
  float dt = endTime - startTime;

  //printf("start time: %f, end time %f, dt: %f, device: %d\n", startTime, endTime, dt, nengoData->device);

  hipError_t err;

  dim3 dimBlock(1, 1);
  dim3 dimGrid(1, 1);

//   jint NDsteps = 
  //float NDadjusted_dt = dt / NDsteps; /// steps;
  jint ND_steps = 1; //(jint)(ceil(dt / nengoData->maxTimeStep));
  float ND_adjusted_dt = dt;// / ND_steps;

  jint steps = 1;
  float adjusted_dt = dt;

//  if(((jint) (startTime * 1000)) < 4)
  //printDynamicNengoGPUData(nengoData);


///////////////////////////////////////////////////////
// Copy input from host to GPU
///////////////////////////////////////////////////////

  hipMemcpy(nengoData->input->array + nengoData->GPUInputSize, sharedInput + nengoData->offsetInSharedInput, (nengoData->JavaInputSize + nengoData->CPUInputSize) * sizeof(float), hipMemcpyHostToDevice);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: copying cpu input to device");

///////////////////////////////////////////////////////
// Multiply input vectors by corresponding termination transform
///////////////////////////////////////////////////////
  dimBlock.x = 256;
  dimGrid.x = nengoData->totalNumTransformRows / dimBlock.x + 1;

  transform<<<dimGrid, dimBlock>>> (dt, nengoData->totalNumTransformRows, nengoData->input->array, nengoData->terminationOffsetInInput->array, nengoData->transformRowToInputIndexor->array, nengoData->terminationTransforms->array, nengoData->terminationTau->array, nengoData->terminationOutput->array, nengoData->terminationOutputIndexor->array, nengoData->inputDimension->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: transform");

///// sum the activation in each dimension of each ensemble

  dimBlock.x = 256;
  dimGrid.x = nengoData->totalEnsembleDimension / dimBlock.x + 1;

  sumTerminations <<<dimGrid, dimBlock>>> (nengoData->totalEnsembleDimension, nengoData->maxNumDecodedTerminations, nengoData->terminationOutput->array, nengoData->ensembleSums->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: sum");


///// process ND (nonDecoded) terminations
  dimBlock.x = 256;
  dimGrid.x = nengoData->numEnsembles / dimBlock.x + 1;

  processNDterminations<<<dimGrid, dimBlock>>>(nengoData->numEnsembles, nengoData->numNDterminations, ND_steps, ND_adjusted_dt, nengoData->NDterminationEnsembleOffset->array, nengoData->terminationOffsetInInput->array, nengoData->inputDimension->array, nengoData->NDterminationInputIndexor->array, nengoData->input->array, nengoData->NDterminationWeights->array, nengoData->NDterminationCurrents->array, nengoData->NDterminationEnsembleSums->array, nengoData->terminationTau->array);

  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: process non decoded");

///// encode
  dimBlock.x = 256;
  dimGrid.x = nengoData->numNeurons / dimBlock.x + 1;

  encode<<<dimGrid, dimBlock>>> (nengoData->numNeurons, nengoData->encoders->array, nengoData->ensembleSums->array, nengoData->encodeResult->array, nengoData->encoderRowToEnsembleIndexor->array, nengoData->ensembleOffsetInDimensions->array, nengoData->ensembleDimension->array, nengoData->encoderStride->array, nengoData->encoderRowToNeuronIndexor->array);


  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: encode");



///// integrate after encoding
  dimBlock.x = 256;
  dimGrid.x = nengoData->numNeurons / dimBlock.x + 1;

    integrateAfterEncode <<<dimGrid, dimBlock>>> (nengoData->numNeurons, dt, adjusted_dt, steps, nengoData->neuronToEnsembleIndexor->array, nengoData->encodeResult->array, nengoData->neuronVoltage->array, nengoData->neuronReftime->array, nengoData->ensembleTauRC->array, nengoData->ensembleTauRef->array, nengoData->neuronBias->array, nengoData->neuronScale->array, nengoData->spikes->array, nengoData->NDterminationEnsembleSums->array, nengoData->isSpikingEnsemble->array);

  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: integrate after encode");

///// decode

  dimBlock.x = 256;
  dimGrid.x = nengoData->totalOutputSize / dimBlock.x + 1;

  decode<<<dimGrid, dimBlock>>>(nengoData->totalOutputSize, nengoData->decoders->array, nengoData->spikes->array, nengoData->ensembleOutput->array, nengoData->decoderRowToEnsembleIndexor->array, nengoData->ensembleNumNeurons->array, nengoData->ensembleOffsetInNeurons->array, nengoData->decoderStride->array, nengoData->decoderRowToOutputIndexor->array);

  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: decode");


//// move output to device

  // reorganize the output, which comes out of decode in terms of ensembles, so that it is in terms of network arrays.
  dimGrid.x = nengoData->totalOutputSize / (dimBlock.x * dimBlock.y) + 1;
  moveGPUData<<<dimGrid, dimBlock>>>(nengoData->totalOutputSize, nengoData->ensembleOutputToNetworkArrayOutputMap->array, nengoData->output->array, nengoData->ensembleOutput->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: moveensembleoutput to network array output");

  if(nengoData->numSpikesToSendBack > 0)
  { 
    dimGrid.x = nengoData->numSpikesToSendBack / (dimBlock.x * dimBlock.y) + 1;
    moveGPUData<<<dimGrid, dimBlock>>>(nengoData->numSpikesToSendBack, nengoData->spikeMap->array, nengoData->output->array + nengoData->totalOutputSize, nengoData->spikes->array);
    err = hipGetLastError();
    checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: extract spikes to send back");
  }

  if(nengoData->CPUOutputSize + nengoData->numSpikesToSendBack > 0)
  {
    hipMemcpy(nengoData->outputHost->array, nengoData->output->array + nengoData->GPUOutputSize, (nengoData->CPUOutputSize + nengoData->numSpikesToSendBack) * sizeof(float), hipMemcpyDeviceToHost);
    err = hipGetLastError();
    checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: move output from GPU to CPU");
  }
  
//// move data along GPU projections
  dimGrid.x = nengoData->GPUInputSize / (dimBlock.x * dimBlock.y) + 1;
  moveGPUData<<<dimGrid, dimBlock>>>(nengoData->GPUInputSize, nengoData->GPUTerminationToOriginMap->array, nengoData->input->array, nengoData->output->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: move output along GPU projections");
}

float* allocateCudaFloatArray(jint size)
{
  float* temp;
  hipError_t err;
  err = hipMalloc((void**)&temp, size * sizeof(float));
  checkCudaError(err, "allocate cuda float array");
  return temp;
}
  
jint* allocateCudaIntArray(jint size)
{
  jint* temp;
  hipError_t err;
  err = hipMalloc((void**)&temp, size * sizeof(jint));
  checkCudaError(err, "allocate cuda jint array");
  return temp;
}

long getDeviceCapacity(jint device)
{
  hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);  
  return deviceProperties.totalGlobalMem;
}
  
void initializeDeviceInputAndOutput(NengoGPUData* nengoData)
{
  char* name;
  hipError_t err;

  name = "input";
  nengoData->input = newFloatArrayOnDevice(nengoData->totalInputSize, name); 
  
  name = "ensembleOutput";
  nengoData->ensembleOutput = newFloatArrayOnDevice(nengoData->totalOutputSize, name); 

  name = "output";
  nengoData->output = newFloatArrayOnDevice(nengoData->totalOutputSize + nengoData->numSpikesToSendBack, name); 
  
  name = "spikes";
  nengoData->spikes = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "terminationOutput";
  nengoData->terminationOutput = newFloatArrayOnDevice(nengoData->totalEnsembleDimension * nengoData->maxNumDecodedTerminations, name); 
  
  name = "ensembleSums";
  nengoData->ensembleSums = newFloatArrayOnDevice(nengoData->totalEnsembleDimension, name); 
  
  name = "encodeResult";
  nengoData->encodeResult = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "neuronVoltage";
  nengoData->neuronVoltage = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "neuronReftime";
  nengoData->neuronReftime = newFloatArrayOnDevice(nengoData->numNeurons, name); 


  err = hipMemset(nengoData->input->array, 0, nengoData->GPUInputSize * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->ensembleOutput->array, 0, nengoData->totalOutputSize * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->output->array, 0, (nengoData->totalOutputSize + nengoData->numSpikesToSendBack) * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->spikes->array, 0,
 nengoData->numNeurons * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->terminationOutput->array, 0, nengoData->totalEnsembleDimension * nengoData->maxNumDecodedTerminations * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->neuronVoltage->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->neuronReftime->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  
  name = "NDterminationCurrents";
  nengoData->NDterminationCurrents = newFloatArrayOnDevice(nengoData->numNDterminations, name); 
  name = "NDterminationEnsembleSum";
  nengoData->NDterminationEnsembleSums = newFloatArrayOnDevice(nengoData->numEnsembles, name); 

  err = hipMemset(nengoData->NDterminationCurrents->array, 0, nengoData->numNDterminations * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->NDterminationEnsembleSums->array, 0, nengoData->numEnsembles * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
}

#ifdef __cplusplus
}
#endif

