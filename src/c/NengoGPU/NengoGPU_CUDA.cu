#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C"{
#endif

#include <stdio.h>
#include <stdlib.h>
//#include <cutil.h>
#include <hip/hip_runtime.h>

#include "NengoGPU.h"
#include "NengoGPU_CUDA.h"

// print the contents of an array of integers located on the device
void printIntArrayFromDevice(FILE* fp, intArray* a, int n, int m)
{
  int* temp = (int*) malloc( m * n * sizeof(int));
  hipMemcpy(temp, a->array, m * n * sizeof(int), hipMemcpyDeviceToHost);

  int i, j;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "line %d: ", i) : printf("line %d:", i);
    for(j = 0; j < n; j++)
    {
      fp ? fprintf(fp, "%d ", temp[i * n + j]) : printf("%d ", temp[i * n + j]);
    }
    fp ? fprintf(fp, "\n") : printf("\n");
  }

  fp ? fprintf(fp, "\n") : printf("\n");

  free(temp);
}

// print the contents of an array of floats located on the device
void printFloatArrayFromDevice(FILE* fp, floatArray* a, int n, int m)
{
  hipError_t err;
  float* temp = (float*) malloc( m * n * sizeof(float));
  err = hipMemcpy(temp, a->array, m * n * sizeof(float), hipMemcpyDeviceToHost);
  checkCudaError(err);

  int i, j;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "line %d: ", i) : printf("line %d:", i);
    for(j = 0; j < n; j++)
    {
      fp ? fprintf(fp, "%f ", temp[i * n + j]) : printf("%f ", temp[i * n + j]);
    }

    fp ? fprintf(fp, "\n") : printf("\n");
  }

  fp ? fprintf(fp, "\n") : printf("\n");

  free(temp);
}

void printIntColumn(FILE* fp, int* array, int m, int n, int col)
{
  int* temp = (int*) malloc( m * n * sizeof(int));
  hipMemcpy(temp, array, m * n * sizeof(int), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "%d ", temp[i * n + col]) : printf("%d ", temp[i * n + col]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

void printFloatColumn(FILE* fp, float* array, int m, int n, int col)
{
  float* temp = (float*) malloc( m * n * sizeof(float));
  hipMemcpy(temp, array, m * n * sizeof(float), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "%f ", temp[i * n + col]) : printf("%f ", temp[i * n + col]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}
 
void printFloatRange(FILE* fp, float* array, int start, int end)
{
  float* temp = (float*) malloc((end - start + 1)  * sizeof(float));
  hipMemcpy(temp, array + start, (end - start + 1) * sizeof(float), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < end - start + 1; i++)
  {
    fp ? fprintf(fp, "%f ", temp[i]) : printf("%f ", temp[i]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

void printIntRange(FILE* fp, int* array, int start, int end)
{
  int* temp = (int*) malloc((end - start + 1)  * sizeof(int));
  hipMemcpy(temp, array + start, (end - start + 1) * sizeof(int), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < end - start + 1; i++)
  {
    fp ? fprintf(fp, "%d ", temp[i]) : printf("%d ", temp[i]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

// get number of devices available
int getGPUDeviceCount(){
  hipError_t err;
  int numDevices;
  
  err = hipGetDeviceCount(&numDevices);
  checkCudaError(err);
  
  return numDevices;
}

// Reserves device with number deviceNum for the thread that calls this function. No interaction with the device should take place until this has been called.
// Once the device is reserved for the thread, no other thread should try to interact with that device or reserve it. A thread can reserve only one device at a time
void initGPUDevice(int deviceNum)
{
  hipSetDevice(deviceNum);
}

void shutdownGPUDevice()
{
}

void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    exit(EXIT_FAILURE);
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalNumTerminationRows kernels are launched.
// Dot product the ith termination row with the corresponding input vector. Integrate the result. Results are stored in terminationValues. 
__global__ void transform(float dt, int numTransformRows, float* input, int* inputOffset, int* transformRowToInputIndexor, float* transforms, float* tau, float* terminationOutput, int* terminationOutputIndexor, int* inputDimensions)
{
  
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < numTransformRows)
  {
    
    int j;
    int inputIndex = transformRowToInputIndexor[i];
    int offset = inputOffset[inputIndex];
    
    int inputDimension = inputDimensions[inputIndex];
    int transformRowIndex = i;
    
    float my_tau = tau[inputIndex];
    
    float dot_product = 0;
    
   for(j=0; j < inputDimension; j++)
    {
      dot_product += input[offset + j] * transforms[transformRowIndex];

      transformRowIndex += numTransformRows;
    }
   
    float dt_over_tau = dt / my_tau;
    

    int outputIndex = terminationOutputIndexor[i];
    terminationOutput[outputIndex] = (1 - dt_over_tau) * terminationOutput[outputIndex] + dt_over_tau * dot_product;
    
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalDimension kernels are launched.
// Sum the termination values for one dimension of one ensemble. Results are stored in ensembleSums.
__global__ void sumTerminations(int totalDimensions, int maxNumDecodedTerminations, float* terminationOutput, float* ensembleSums)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < totalDimensions)
  {
    int terminationOutputIndex = i;
    int j;
    float sum = 0;

    for(j=0; j < maxNumDecodedTerminations; j++)
    {
      sum += terminationOutput[terminationOutputIndex];
      terminationOutputIndex += totalDimensions;
    }

    ensembleSums[i] = sum;
  }
}

extern __shared__ char shared[];
// Kernel, run on GPU. block size and grid size should be set so that at least numNeurons kernels are launched.
// Multiply one encoder row by the sum vector for the corresponding ensemble. Then integrate to determine whether the neuron corresponding to that encoder row should spike. Results stored in spikes.
__global__ void encode(int maxDimension, float* encoders, float* sums, float* encodingResult, int* blockToEnsembleMap, int* ensembleIndexOfFirstBlock, int* ensembleOffsetInDimension, int* ensembleOffsetInNeurons, int* ensembleOffsetInEncoders, int* ensembleNumNeurons, int* ensembleDimension, int* encoderStride)
{
  int thread_id = threadIdx.x;// + blockDim.x * threadIdx.y;
  int block_size = blockDim.x * blockDim.y;
  int block_id = blockIdx.x;// + gridDim.x * blockIdx.y;
  
  int ensembleIndex = blockToEnsembleMap[block_id];
  int numNeurons = ensembleNumNeurons[ensembleIndex];
  int ensembleFirstBlock = ensembleIndexOfFirstBlock[ensembleIndex];
  int dimension = ensembleDimension[ensembleIndex];
  int neuronIndexInEnsemble = thread_id + (block_id - ensembleFirstBlock) * block_size;
  int dimensionOffset = ensembleOffsetInDimension[ensembleIndex];


  int j, index;
  // now load the the sum vector and encoder stride vector into shared memory
  int* encoder_stride_shared = (int*)shared;
  float* sum_shared = (float*)(encoder_stride_shared + maxDimension);

  for(j = 0; j < dimension; j+=block_size)
  {
    index = j + thread_id;

    if(index < maxDimension)
      encoder_stride_shared[index] = encoderStride[index];
  }

  syncthreads();


  for(j = 0; j < dimension; j+=block_size)
  {
    index = j + thread_id;

    if(index < dimension)
      sum_shared[index] = sums[dimensionOffset + index];
  }

  syncthreads();


  if( neuronIndexInEnsemble < numNeurons)
  {
    int neuronIndexOnDevice = neuronIndexInEnsemble + ensembleOffsetInNeurons[ensembleIndex];
    int encoderOffset = neuronIndexInEnsemble + ensembleOffsetInEncoders[ensembleIndex];
    float dot_product = 0;
    for(j=0; j < dimension; j++)
    {
      dot_product += encoders[encoderOffset] * sum_shared[j];
      encoderOffset += encoder_stride_shared[j];
    }

    encodingResult[neuronIndexOnDevice] = dot_product;
  }
  
}

__global__ void integrateAfterEncode(int numNeurons, float dt, float adjusted_dt, int steps, int* neuronToEnsembleIndexor, float* encodingResult, float* neuronVoltage, float* neuronReftime, float* tau_RC, float* tauRef, float* bias, float* scale, float* spikes, float* NDterminationSums)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;
  
  if( i < numNeurons)
  {
    int ensembleIndex = neuronToEnsembleIndexor[i];
    float voltage = neuronVoltage[i];
    float refTime = neuronReftime[i];
    float tau_rc = tau_RC[ensembleIndex];
    float tau_ref = tauRef[ensembleIndex];
    float current = bias[i] + scale[i] * (encodingResult[i] + NDterminationSums[ensembleIndex]);
    float dV, post_ref, v_threshold = 1.0f;
    float spike_float;
    int j, spike = 0;

    for(j = 0; j < steps; j++)
    {
      dV = adjusted_dt / tau_rc * (current - voltage);
      voltage = max(voltage + dV, 0.0f);

      post_ref = 1.0f - (refTime - adjusted_dt) / adjusted_dt;

      voltage = post_ref >= 1.0f ? voltage : voltage * post_ref;

      voltage = post_ref <= 0.0f ? 0.0f : voltage;

      spike = spike ? spike : voltage > v_threshold;
      spike_float = spike ? 1.0f/dt : 0.0f;
      refTime = spike ? ((adjusted_dt / dV) * (dV - voltage + v_threshold)) + tau_ref : refTime - adjusted_dt;
      voltage = spike ? 0.0 : voltage;
    }

    neuronReftime[i] = refTime;
    neuronVoltage[i] = voltage;
    spikes[i] = spike_float;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalOutputSize kernels are launched.
// Multiply one decoder row by the spike vector for the corresponding ensemble. The result is one dimension of the output vector for the ensemble. Results stored in output.
__global__ void decode(int maxNumNeurons, int* blockToEnsembleMap, int* ensembleNumNeurons, int* ensembleIndexOfFirstBlock, int* ensembleOffsetInNeurons, int* ensembleOutputSize, float* spikes, float* decoders, float* output, int* ensembleOffsetInDecoders, int* ensembleOffsetInOutput, int* decoderStride)
{
  
  int thread_id = threadIdx.x;// + blockDim.x * threadIdx.y;
  int block_size = blockDim.x * blockDim.y;
  int block_id = blockIdx.x ;//+ gridDim.x * blockIdx.y;

  int ensembleIndex = blockToEnsembleMap[block_id];
  int numNeurons = ensembleNumNeurons[ensembleIndex];
  int ensembleFirstBlock = ensembleIndexOfFirstBlock[ensembleIndex];
  int outputIndexInEnsemble = thread_id + (block_id - ensembleFirstBlock) * block_size;
  int spikesOffset = ensembleOffsetInNeurons[ensembleIndex];
  int outputSize = ensembleOutputSize[ensembleIndex];
  
  
  int j, index;
  int* decoder_stride_shared = (int*)shared;
  float* spikes_shared = (float*)(decoder_stride_shared + maxNumNeurons);

  for(j = 0; j < numNeurons; j+=block_size)
  {
    index = j + thread_id;

    if(index < numNeurons)
      decoder_stride_shared[index] = decoderStride[index];
  }

  syncthreads();
  
  for(j = 0; j < numNeurons; j+=block_size)
  {
    index = j + thread_id;

    if(index < numNeurons)
      spikes_shared[index] = spikes[spikesOffset + index];
  }
  
  syncthreads();


  if(outputIndexInEnsemble < outputSize)
  {
    
    int decoderOffset = outputIndexInEnsemble + ensembleOffsetInDecoders[ensembleIndex]; 
    float dot_product = 0;
    for(j=0; j < numNeurons; j++)
    {
      dot_product += decoders[decoderOffset] * spikes_shared[j];

      decoderOffset += decoder_stride_shared[j];
    }

    int outputIndexOnDevice = outputIndexInEnsemble + ensembleOffsetInOutput[ensembleIndex];
    output[outputIndexOnDevice] = dot_product;
  }
}



// launch as many as there are ensembles
__global__ void processNDterminations(int numEnsembles, int numNDterminations, int steps, float adjusted_dt, int* NDterminationEnsembleOffset, int* inputOffsets, int* inputIndex, float* input, float* weights, float* current, float* sum, float* tau)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < numEnsembles)
  {
    int offset = NDterminationEnsembleOffset[i];
    int count = (i == numEnsembles - 1) ? numNDterminations - offset : NDterminationEnsembleOffset[i+1] - offset;
    int j, inputOffset, index;
    float val, temp_sum = 0, temp_current, temp_tau;

    for(j = 0; j < count; j++)
    {
      index = inputIndex[offset + j];
      inputOffset = inputOffsets[index]; 

      val = input[inputOffset] * weights[offset + j];
      temp_current = current[offset + j];
      temp_tau = tau[index];

      for(j = 0; j < steps; j++)
      {
        //temp_current = (temp_current + val * adjusted_dt / temp_tau) * (1 - adjusted_dt / temp_tau);
        temp_current += val * adjusted_dt / temp_tau;
        temp_current *= (1 - adjusted_dt / temp_tau);
      }

      current[offset + j] = temp_current;
      //current[offset + j] = temp_tau;
      
      temp_sum += temp_current;
    }

    sum[i] = temp_sum;
  }
}


__global__ void moveGPUOutputIntoInput(int GPUInputSize, int* map, float* input, float* output)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < GPUInputSize)
  {
    input[ i ] = output[ map[i] ];
  }
}
      
// run a NengoGPUData struct for one step
void run_NEFEnsembles(NengoGPUData* nengoData, float startTime, float endTime)
{
  float dt = endTime - startTime;

  //printf("start time: %f, end time %f, dt: %f\n", startTime, endTime, dt);

  hipError_t err;

  dim3 dimBlock(1, 1);
  dim3 dimGrid(1, 1);

  int steps = (int)(ceil(dt / nengoData->maxTimeStep));
  float adjusted_dt = (int) (dt / steps); /// steps;



///////////////////////////////////////////////////////
// Copy input from host to GPU
///////////////////////////////////////////////////////

 // printf("Copy input from host\n");
  err = hipMemcpy(nengoData->input->array + nengoData->GPUInputSize, nengoData->inputHost->array, nengoData->CPUInputSize * sizeof(float), hipMemcpyHostToDevice);
  err = hipGetLastError();
  checkCudaError(err);


///////////////////////////////////////////////////////
// Multiply input vectors by corresponding termination transform
///////////////////////////////////////////////////////
  dimBlock.x = 256;
  dimGrid.x = nengoData->totalNumTransformRows / dimBlock.x + 1;

//  printf("transform\n");
  transform<<<dimGrid, dimBlock>>> (dt, nengoData->totalNumTransformRows, nengoData->input->array, nengoData->inputOffset->array, nengoData->transformRowToInputIndexor->array, nengoData->terminationTransforms->array, nengoData->terminationTau->array, nengoData->terminationOutput->array, nengoData->terminationOutputIndexor->array, nengoData->inputDimensions->array);
  err = hipGetLastError();
  checkCudaError(err);

///// sum

  dimBlock.x = 256;
  dimGrid.x = nengoData->totalEnsembleDimension / dimBlock.x + 1;

  //printf("sum\n");
  sumTerminations <<<dimGrid, dimBlock>>> (nengoData->totalEnsembleDimension, nengoData->maxNumDecodedTerminations, nengoData->terminationOutput->array, nengoData->ensembleSums->array);
  err = hipGetLastError();
  checkCudaError(err);


  //printf("ensembleSums:\n");

///// process ND (nonDecoded) terminations
  dimBlock.x = 256;
  dimGrid.x = nengoData->numEnsembles / dimBlock.x + 1;

  //printf("process ND\n");
  processNDterminations<<<dimGrid, dimBlock>>>(nengoData->numEnsembles, nengoData->numNDterminations, 1, dt, nengoData->NDterminationEnsembleOffset->array, nengoData->inputOffset->array, nengoData->NDterminationInputIndexor->array, nengoData->input->array, nengoData->NDterminationWeights->array, nengoData->NDterminationCurrents->array, nengoData->NDterminationEnsembleSums->array, nengoData->terminationTau->array);

  err = hipGetLastError();
  checkCudaError(err);



///// encode
  int cuda_sharedMemSize = 2 * nengoData->maxDimension * sizeof(int);

  dimBlock.x = nengoData->blockSizeForEncode;
  dimGrid.x = nengoData->numBlocksForEncode;

//  printf("encode\n");
  encode<<<dimGrid, dimBlock, cuda_sharedMemSize>>> (nengoData->maxDimension, nengoData->encoders->array, nengoData->ensembleSums->array, nengoData->encodeResult->array, nengoData->blockToEnsembleMapForEncode->array, nengoData->ensembleIndexOfFirstBlockForEncode->array, nengoData->ensembleOffsetInDimensions->array, nengoData->ensembleOffsetInNeurons->array, nengoData->ensembleOffsetInEncoders->array, nengoData->ensembleNumNeurons->array, nengoData->ensembleDimension->array, nengoData->encoderStride->array);

  err = hipGetLastError();
  checkCudaError(err);



///// integrate after encoding
  dimBlock.x = 256;
  dimGrid.x = nengoData->numNeurons / dimBlock.x + 1;

//  printf("integrate after encode\n");
  integrateAfterEncode <<<dimGrid, dimBlock>>> (nengoData->numNeurons, dt, dt, 1, nengoData->neuronToEnsembleIndexor->array, nengoData->encodeResult->array, nengoData->neuronVoltage->array, nengoData->neuronReftime->array, nengoData->ensembleTauRC->array, nengoData->ensembleTauRef->array, nengoData->neuronBias->array, nengoData->neuronScale->array, nengoData->spikes->array, nengoData->NDterminationEnsembleSums->array);

  err = hipGetLastError();
  checkCudaError(err);


///// decode
  dimBlock.x = nengoData->blockSizeForDecode;
  dimGrid.x = nengoData->numBlocksForDecode;

  cuda_sharedMemSize = 2 * nengoData->maxNumNeurons * sizeof(int);


 // printf("decode\n");
  decode <<<dimGrid, dimBlock, cuda_sharedMemSize>>> (nengoData->maxNumNeurons, nengoData->blockToEnsembleMapForDecode->array, nengoData->ensembleNumNeurons->array, nengoData->ensembleIndexOfFirstBlockForDecode->array, nengoData->ensembleOffsetInNeurons->array, nengoData->ensembleOutputSize->array, nengoData->spikes->array, nengoData->decoders->array, nengoData->output->array, nengoData->ensembleOffsetInDecoders->array, nengoData->ensembleOffsetInOutput->array, nengoData->decoderStride->array);

  err = hipGetLastError();
  checkCudaError(err);


//// move output to device

  //printf("copy output from device\n");
  hipMemcpy(nengoData->outputHost->array, nengoData->output->array, nengoData->totalOutputSize * sizeof(float), hipMemcpyDeviceToHost);
  err = hipGetLastError();
  checkCudaError(err);
  

  //printf("copy spikes from device\n");
  hipMemcpy(nengoData->spikesHost->array, nengoData->spikes->array, nengoData->numNeurons * sizeof(float), hipMemcpyDeviceToHost);
  err = hipGetLastError();
  checkCudaError(err);

//// move data along GPU projections
  dimGrid.y = nengoData->totalOutputSize / (dimBlock.x * dimBlock.y) + 1;
  //printf("move output along projections\n");
  moveGPUOutputIntoInput<<<dimGrid, dimBlock>>>(nengoData->GPUInputSize, nengoData->GPUTerminationToOriginMap->array, nengoData->input->array, nengoData->output->array);
  err = hipGetLastError();
  checkCudaError(err);
}

float* allocateCudaFloatArray(int size)
{
  float* temp;
  hipError_t err;
  err = hipMalloc((void**)&temp, size * sizeof(float));
  checkCudaError(err);
  return temp;
}
  
int* allocateCudaIntArray(int size)
{
  int* temp;
  hipError_t err;
  err = hipMalloc((void**)&temp, size * sizeof(int));
  checkCudaError(err);
  return temp;
}
  
void initializeDeviceInputAndOutput(NengoGPUData* nengoData)
{
  char* name;
  hipError_t err;

  name = "input";
  nengoData->input = newFloatArrayOnDevice(nengoData->totalInputSize, name); 
  
  name = "output";
  nengoData->output = newFloatArrayOnDevice(nengoData->totalOutputSize, name); 
  
  name = "spikes";
  nengoData->spikes = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "terminationOutput";
  nengoData->terminationOutput = newFloatArrayOnDevice(nengoData->totalEnsembleDimension * nengoData->maxNumDecodedTerminations, name); 
  
  name = "ensembleSums";
  nengoData->ensembleSums = newFloatArrayOnDevice(nengoData->totalEnsembleDimension, name); 
  
  name = "encodeResult";
  nengoData->encodeResult = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "neuronVoltage";
  nengoData->neuronVoltage = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "neuronReftime";
  nengoData->neuronReftime = newFloatArrayOnDevice(nengoData->numNeurons, name); 


  err = hipMemset(nengoData->input->array, 0, nengoData->GPUInputSize * sizeof(float));
  checkCudaError(err);
  err = hipMemset(nengoData->output->array, 0, nengoData->totalOutputSize * sizeof(float));
  checkCudaError(err);
  err = hipMemset(nengoData->spikes->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaError(err);
  err = hipMemset(nengoData->terminationOutput->array, 0, nengoData->totalEnsembleDimension * nengoData->maxNumDecodedTerminations * sizeof(float));
  checkCudaError(err);
  err = hipMemset(nengoData->neuronVoltage->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaError(err);
  err = hipMemset(nengoData->neuronReftime->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaError(err);
  
  name = "NDterminationCurrents";
  nengoData->NDterminationCurrents = newFloatArrayOnDevice(nengoData->numNDterminations, name); 
  name = "NDterminationEnsembleSum";
  nengoData->NDterminationEnsembleSums = newFloatArrayOnDevice(nengoData->numEnsembles, name); 

  err = hipMemset(nengoData->NDterminationCurrents->array, 0, nengoData->numNDterminations * sizeof(float));
  checkCudaError(err);
  err = hipMemset(nengoData->NDterminationEnsembleSums->array, 0, nengoData->numEnsembles * sizeof(float));
  checkCudaError(err);
}

#ifdef __cplusplus
}
#endif

