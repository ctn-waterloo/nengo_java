#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C"{
#endif

#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <hip/hip_runtime.h>

#include "NengoGPU.h"
#include "NengoGPU_CUDA.h"

// print the contents of an array of integers located on the device
void printIntArrayFromDevice(FILE* fp, int* array, int size)
{
  int* temp = (int*) malloc( size * sizeof(int));
  hipMemcpy(temp, array, size * sizeof(int), hipMemcpyDeviceToHost);

  int i = 0;
  for(; i < size; i++)
  {
    if(fp != NULL)
    {
      fprintf(fp, "%d ", temp[i]);
    }
    else
    {
      printf("%d ", temp[i]);
    }
  }

  if(fp != NULL)
  {
    fprintf(fp, "\n");
  }
  else
  {
    printf("\n");
  }

  free(temp);
}

// print the contents of an array of floats located on the device
void printFloatArrayFromDevice(FILE* fp, float* array, int size)
{
  float* temp = (float*) malloc( size * sizeof(float));
  hipMemcpy(temp, array, size * sizeof(float), hipMemcpyDeviceToHost);

  int i = 0;
  for(; i < size; i++)
  {
    if(fp != NULL)
    {
      fprintf(fp, "%f ", temp[i]);
    }
    else
    {
      printf("%f ", temp[i]);
    }
  }

  if(fp != NULL)
  {
    fprintf(fp, "\n");
  }
  else
  {
    printf("\n");
  }

  free(temp);
}


// get number of devices available
int getGPUDeviceCount(){
  hipError_t err;
  int numDevices;
  
  err = hipGetDeviceCount(&numDevices);
  checkCudaError(err);
  
  return numDevices;
}

// Reserves device with number deviceNum for the thread that calls this function. No interaction with the device should take place until this has been called.
// Once the device is reserved for the thread, no other thread should try to interact with that device or reserve it. A thread can reserve only one device at a time
void initGPUDevice(int deviceNum)
{
  hipSetDevice(deviceNum);
}

void shutdownGPUDevice()
{
}

void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    exit(EXIT_FAILURE);
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalNumTerminationRows kernels are launched.
// Dot product the ith termination row with the corresponding input vector. Integrate the result. Results are stored in terminationValues. 
__global__ void transformAndIntegrate(float dt, int totalNumTerminationRows, float* input, float* transforms, float* terminationTauValues, float* terminationValues, int* inputIndices, int* transformIndices, int* terminationRowToTerminationIndexor, int* terminationDimensions)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < totalNumTerminationRows)
  {
    int terminationIndex = terminationRowToTerminationIndexor[i];
    int terminationDimension = terminationDimensions[terminationIndex];

    int j = 0;
    int inputIndex = terminationIndex;
    int transformIndex = i;
  
    float dot_product = 0;
    for(; j < terminationDimension; j++)
    {
      dot_product += input[inputIndex] * transforms[transformIndex];

      transformIndex = transformIndices[transformIndex];

      inputIndex = inputIndices[inputIndex];
    }

    float dt_over_tau = dt / terminationTauValues[terminationIndex];

    terminationValues[i] = (1 - dt_over_tau) * terminationValues[i] + dt_over_tau * dot_product;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalDimension kernels are launched.
// Sum the termination values for one dimension of one ensemble. Results are stored in ensembleSums.
__global__ void sumTerminations(int totalDimensions, float* ensembleSums, float* terminationValues, int* dimensionToEnsembleIndexor, int* dimensionIndexInEnsemble, int* ensembleDimensions, int* ensembleTerminations, int* ensemblePositionsInTerminationValues, int* sumHelper)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < totalDimensions)
  {
  int ensembleIndex = dimensionToEnsembleIndexor[i];
  int ensembleDimension = ensembleDimensions[ensembleIndex];
  int ensembleNumTerminations = ensembleTerminations[ensembleIndex];
  int indexInEnsemble = dimensionIndexInEnsemble[i];
  int ensembleStart = ensemblePositionsInTerminationValues[ensembleIndex];

  int j = 0;
  float sum = 0;

  for(; j < ensembleNumTerminations; j++)
  {
    sum += terminationValues[ensembleStart + ensembleDimension * j + indexInEnsemble];
  }

  ensembleSums[sumHelper[i]] = sum;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least numNeurons kernels are launched.
// Multiply one encoder row by the sum vector for the corresponding ensemble. Then integrate to determine whether the neuron corresponding to that encoder row should spike. Results stored in spikes.
__global__ void encodeAndIntegrate(float dt, float adjusted_dt, int steps, int totalNumNeurons, float* encoders, float* ensembleSums, int* encoderIndices, int* sumIndices, int* ensembleDimensions, float* neuronVoltage, float* neuronReftime, float* spikes, int* spikesHelper, int* neuronToEnsembleIndexor, float* ensembleTauRC, float* ensembleTauRef, float* bias, float* scale)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < totalNumNeurons)
  {

  int ensembleIndex = neuronToEnsembleIndexor[i];
  int sumIndex = ensembleIndex;
  int encoderIndex = i;
  int ensembleDimension = ensembleDimensions[ensembleIndex];
  
  int j = 0;
  float dot_product = 0;
  for(; j < ensembleDimension; j++)
  {
    dot_product += encoders[encoderIndex] * ensembleSums[sumIndex];

    encoderIndex = encoderIndices[encoderIndex];
    sumIndex = sumIndices[sumIndex];
  }

  // integrate
  float voltage = neuronVoltage[i];
  float refTime = neuronReftime[i];
  float tau_rc = ensembleTauRC[ensembleIndex];
  float tau_ref = ensembleTauRef[ensembleIndex];
  float current = bias[i] + scale[i] * dot_product;
  float dV, post_ref, v_threshold = 1.0f;
  float spike_float;
  int spikeIndex, spike = 0;
  

  for(j = 0; j < steps; j++)
  {
    dV = adjusted_dt / tau_rc * (current - voltage);
    voltage = max(voltage + dV, 0.0f);

    post_ref = 1.0f - (refTime - adjusted_dt) / adjusted_dt;

    voltage = post_ref >= 1.0f ? voltage : voltage * post_ref;

    voltage = post_ref <= 0.0f ? 0.0f : voltage;

    v_threshold = 1.0f;

    spike = spike ? spike : voltage > v_threshold;
    spike_float = spike ? 1.0f/dt : 0.0f;
    refTime = spike ? ((adjusted_dt / dV) * (dV - voltage + v_threshold)) + tau_ref : refTime - adjusted_dt;
    voltage = spike ? 0.0 : voltage;
  }

  neuronReftime[i] = refTime;
  neuronVoltage[i] = voltage;
  spikeIndex = spikesHelper[i];
  spikes[spikeIndex] = spike_float;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalOutputSize kernels are launched.
// Multiply one decoder row by the spike vector for the corresponding ensemble. The result is one dimension of the output vector for the ensemble. Results stored in output.
__global__ void decode(int totalOutputSize, float* decoder, float* spikes, int* ensembleNumNeurons, float* output, int* decoderIndices, int* spikeIndices, int* decoderRowToEnsembleIndexor)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < totalOutputSize)
  {
  int ensembleIndex = decoderRowToEnsembleIndexor[i];
  int decoderIndex = i;
  int spikeIndex = ensembleIndex;
  int numNeurons = ensembleNumNeurons[ensembleIndex];

  int j = 0;
  float dot_product = 0;

  for(; j < numNeurons; j++)
  {
    dot_product += decoder[decoderIndex] * spikes[spikeIndex];

    decoderIndex = decoderIndices[decoderIndex];

    spikeIndex = spikeIndices[spikeIndex];
  }

  output[i] = dot_product;
  }
}

// run a NengoGPUData object for one step
void run_NEFEnsembles(NengoGPUData* nengoData, float startTime, float endTime)
{
  float dt = endTime - startTime;
  hipError_t err;

  dim3 dimBlock(16, 16);
  dim3 dimGrid(1, nengoData->totalNumTransformRows / (dimBlock.x * dimBlock.y) + 1);

  nengoData->input = moveToDeviceFloatArray(nengoData->input, nengoData->totalInputSize);

  transformAndIntegrate <<<dimGrid, dimBlock>>> (dt, nengoData->totalNumTransformRows, nengoData->input, nengoData->terminationTransforms, nengoData->terminationTau, nengoData->terminationValues, nengoData->inputIndices, nengoData-> terminationTransformIndices, nengoData->terminationRowToTerminationIndexor, nengoData->terminationDimensions);
  err = hipGetLastError();
  checkCudaError(err);
  
  err = hipFree(nengoData->input);
  checkCudaError(err);

  nengoData->input = (float*)malloc(nengoData->totalInputSize * sizeof(float));
  if(!nengoData->input)
  {
    printf("bad malloc\n");
    exit(EXIT_FAILURE);
  }


  dimGrid.y = nengoData->totalDimension / (dimBlock.x * dimBlock.y) + 1;
  sumTerminations <<<dimGrid, dimBlock>>> (nengoData->totalDimension, nengoData->ensembleSums, nengoData->terminationValues, nengoData->dimensionToEnsembleIndexor, nengoData->dimensionIndexInEnsemble, nengoData->ensembleDimensions, nengoData->ensembleTerminations, nengoData->ensemblePositionsInTerminationValues, nengoData->sumHelper);
  err = hipGetLastError();
  checkCudaError(err);

  dimGrid.y = nengoData->numNeurons / (dimBlock.x * dimBlock.y) + 1;
  

  int encoder_steps = (int)ceil(dt / nengoData->maxTimeStep);
  //float encoder_dt = dt / encoder_steps;

  encodeAndIntegrate <<<dimGrid, dimBlock>>> (dt, dt, 1, nengoData->numNeurons, nengoData->encoders, nengoData->ensembleSums, nengoData->encoderIndices, nengoData->sumIndices, nengoData->ensembleDimensions, nengoData->neuronVoltage, nengoData->neuronReftime, nengoData->spikes, nengoData->spikeHelper, nengoData->neuronToEnsembleIndexor, nengoData->ensembleTauRC, nengoData->ensembleTauRef, nengoData->neuronBias, nengoData->neuronScale);
  err = hipGetLastError();
  checkCudaError(err);
  

  if(nengoData->output)
  {
    free(nengoData->output);
  }
    
  err = hipMalloc((void**)&nengoData->output, nengoData->totalOutputSize * sizeof(float)); 
  checkCudaError(err);

  dimGrid.y = nengoData->totalOutputSize / (dimBlock.x * dimBlock.y) + 1;
  decode <<<dimGrid, dimBlock>>> (nengoData->totalOutputSize, nengoData->decoders, nengoData->spikes, nengoData->ensembleNeurons, nengoData->output, nengoData->decoderIndices, nengoData->spikeIndices, nengoData->decoderRowToEnsembleIndexor);
  err = hipGetLastError();
  checkCudaError(err);

  nengoData->output = moveToHostFloatArray(nengoData->output, nengoData->totalOutputSize);

  hipMemcpy(nengoData->spikesHost, nengoData->spikes, nengoData->numNeurons * sizeof(float), hipMemcpyDeviceToHost);
}

#ifdef __cplusplus
}
#endif

